#include "hip/hip_runtime.h"
#include <iostream>

#include "GL\glew.h"
#include "GLFW\glfw3.h"

#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
//#include <helper_cuda_gl.h>

#include "memTransfer.h"

using namespace std;

__global__ void memTransferCuda(hipTextureObject_t src, void *dst, unsigned int wid, unsigned int ht) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Get texture value and write to global memory
	((GLubyte*)dst)[y * wid + x] = tex2D<GLubyte>(src, x, y);
}

void memTransfer(hipArray_t srcArray, void* dst, size_t width, size_t height) {
	// Allocate pitched linear memory
	int fmtSize = sizeof(GLubyte);
	size_t pitch;
	cout << "Format size: " << fmtSize << endl;
	checkCudaErrors(hipMalloc(&dst, width * fmtSize * 4 * height));
	//checkCudaErrors(hipMallocPitch(&dst, &pitch, width * fmtSize * 4, height));
	//cout << "Alloc size: " << pitch * extent.height << endl;
	//cout << "Pitch: " << pitch << endl;


	// Check if the pointers are on device or host
	hipPointerAttribute_t attr;
	checkCudaErrors(hipPointerGetAttributes(&attr, dst));
	cout << "Dst attributes: " << attr.devicePointer << " " << attr.hostPointer << " " << attr.type << endl;
	checkCudaErrors(hipPointerGetAttributes(&attr, srcArray));
	cout << "Source attributes: " << attr.devicePointer << " " << attr.hostPointer << " " << attr.type << endl;

	// Memcpy or Memset
	checkCudaErrors(hipMemcpy2DFromArray(dst, width * fmtSize * 4, srcArray, 0, 0, width * fmtSize * 4, height, hipMemcpyDefault));
	//cout << "Completed memory transfer" << endl;
	//checkCudaErrors(hipMemset2D(pycuda_tex, pitch, 100, extent.width * fmtSize * 4, extent.height));
	//checkCudaErrors(hipFree(pycuda_tex));


	//// Create texture object
	//hipTextureObject_t texObj = 0;

	//hipResourceDesc resDesc;
	//memset(&resDesc, 0, sizeof(resDesc));
	//resDesc.resType = hipResourceTypeArray;
	//resDesc.res.array.array = srcArray;

	//hipTextureDesc texDesc;
	//memset(&texDesc, 0, sizeof(texDesc));
	//texDesc.addressMode[0] = hipAddressModeClamp;
	//texDesc.addressMode[1] = hipAddressModeClamp;
	//texDesc.filterMode = hipFilterModePoint;
	//texDesc.readMode = hipReadModeElementType;
	//texDesc.normalizedCoords = false;

	//hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	//cout << "Created texture object" << endl;
 //   
	//dim3 dimBlock(16, 16);
	//dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
 //   memTransferCuda <<< dimGrid, dimBlock >>> (texObj, dst, width, height);
}